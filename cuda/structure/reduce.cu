#include "hip/hip_runtime.h"
//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "cuda_util.h"
#include "cuda_context.h"
#include "reduce.h"
#include "temp_buffer.h"

#define THRUST_IGNORE_CUB_VERSION_CHECK
#include <cub/device/device_reduce.cuh>
#include <cub/iterator/counting_input_iterator.cuh>

namespace {

template<typename T>
__global__ void cwise_mult_kernel(int len, int stride_a, int stride_b, const T *a, const T *b, T *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= len)
        return;
    out[i] = a[i * stride_a] * b[i * stride_b];
}

/// Custom iterator for allowing strided access with CUB
template<typename T>
struct cub_strided_iterator {
    typedef cub_strided_iterator<T> self_type;
    typedef std::ptrdiff_t difference_type;
    typedef T value_type;
    typedef T *pointer;
    typedef T &reference;

    typedef std::random_access_iterator_tag iterator_category;///< The iterator category

    T *ptr = nullptr;
    int stride = 1;

    __device__ self_type operator++(int) {
        return ++(self_type(*this));
    }

    __device__ self_type &operator++() {
        ptr += stride;
        return *this;
    }

    __host__ __device__ __forceinline__ reference operator*() const {
        return *ptr;
    }

    __device__ self_type operator+(difference_type n) const {
        return self_type(*this) += n;
    }

    __device__ self_type &operator+=(difference_type n) {
        ptr += n * stride;
        return *this;
    }

    __device__ self_type operator-(difference_type n) const {
        return self_type(*this) -= n;
    }

    __device__ self_type &operator-=(difference_type n) {
        ptr -= n * stride;
        return *this;
    }

    __device__ difference_type operator-(const self_type &other) const {
        return (ptr - other.ptr) / stride;
    }

    __device__ reference operator[](difference_type n) const {
        return *(ptr + n * stride);
    }

    __device__ pointer operator->() const {
        return ptr;
    }

    __device__ bool operator==(const self_type &rhs) const {
        return (ptr == rhs.ptr);
    }

    __device__ bool operator!=(const self_type &rhs) const {
        return (ptr != rhs.ptr);
    }
};

template<typename T>
void array_sum_device(const T *ptr_a, T *ptr_out, int count, int byte_stride, int type_length) {
    assert((byte_stride % sizeof(T)) == 0);
    const int stride = byte_stride / sizeof(T);

    void *context = cuda_context_get_current();
    TemporaryBuffer &cub_temp = g_temp_buffer_map[context];

    ContextGuard guard(context);
    auto stream = static_cast<hipStream_t>(cuda_stream_get_current());

    cub_strided_iterator<const T> ptr_strided{ptr_a, stride};

    size_t buff_size = 0;
    check_cuda(hipcub::DeviceReduce::Sum(nullptr, buff_size, ptr_strided, ptr_out, count, stream));
    cub_temp.ensure_fits(buff_size);

    for (int k = 0; k < type_length; ++k) {
        cub_strided_iterator<const T> ptr_strided{ptr_a + k, stride};
        check_cuda(hipcub::DeviceReduce::Sum(cub_temp.buffer, buff_size, ptr_strided, ptr_out + k, count, stream));
    }
}

template<typename T>
void array_sum_device_dispatch(const T *ptr_a, T *ptr_out, int count, int byte_stride, int type_length) {
    using vec2 = wp::vec_t<2, T>;
    using vec3 = wp::vec_t<3, T>;
    using vec4 = wp::vec_t<4, T>;

    // specialized calls for common vector types

    if ((type_length % 4) == 0 && (byte_stride % sizeof(vec4)) == 0) {
        return array_sum_device(reinterpret_cast<const vec4 *>(ptr_a), reinterpret_cast<vec4 *>(ptr_out), count,
                                byte_stride, type_length / 4);
    }

    if ((type_length % 3) == 0 && (byte_stride % sizeof(vec3)) == 0) {
        return array_sum_device(reinterpret_cast<const vec3 *>(ptr_a), reinterpret_cast<vec3 *>(ptr_out), count,
                                byte_stride, type_length / 3);
    }

    if ((type_length % 2) == 0 && (byte_stride % sizeof(vec2)) == 0) {
        return array_sum_device(reinterpret_cast<const vec2 *>(ptr_a), reinterpret_cast<vec2 *>(ptr_out), count,
                                byte_stride, type_length / 2);
    }

    return array_sum_device(ptr_a, ptr_out, count, byte_stride, type_length);
}

template<typename T>
__device__ T element_inner_product(const T &a, const T &b) {
    return a * b;
}

template<unsigned Length, typename T>
__device__ T element_inner_product(const wp::vec_t<Length, T> &a, const wp::vec_t<Length, T> &b) {
    return wp::dot(a, b);
}

/// Custom iterator for allowing strided access with CUB
template<typename ElemT, typename ScalarT>
struct cub_inner_product_iterator {
    typedef cub_inner_product_iterator<ElemT, ScalarT> self_type;
    typedef std::ptrdiff_t difference_type;
    typedef ScalarT value_type;
    typedef ScalarT *pointer;
    typedef ScalarT reference;

    typedef std::random_access_iterator_tag iterator_category;///< The iterator category

    const ElemT *ptr_a = nullptr;
    const ElemT *ptr_b = nullptr;

    int stride_a = 1;
    int stride_b = 1;
    int type_length = 1;

    __device__ self_type operator++(int) {
        return ++(self_type(*this));
    }

    __device__ self_type &operator++() {
        ptr_a += stride_a;
        ptr_b += stride_b;
        return *this;
    }

    __host__ __device__ __forceinline__ reference operator*() const {
        return compute_value(0);
    }

    __device__ self_type operator+(difference_type n) const {
        return self_type(*this) += n;
    }

    __device__ self_type &operator+=(difference_type n) {
        ptr_a += n * stride_a;
        ptr_b += n * stride_b;
        return *this;
    }

    __device__ self_type operator-(difference_type n) const {
        return self_type(*this) -= n;
    }

    __device__ self_type &operator-=(difference_type n) {
        ptr_a -= n * stride_a;
        ptr_b -= n * stride_b;
        return *this;
    }

    __device__ difference_type operator-(const self_type &other) const {
        return (ptr_a - other.ptr_a) / stride_a;
    }

    __device__ reference operator[](difference_type n) const {
        return compute_value(n);
    }

    __device__ bool operator==(const self_type &rhs) const {
        return (ptr_a == rhs.ptr_a);
    }

    __device__ bool operator!=(const self_type &rhs) const {
        return (ptr_a != rhs.ptr_a);
    }

private:
    __device__ ScalarT compute_value(difference_type n) const {
        ScalarT val(0);
        const ElemT *a = ptr_a + n * stride_a;
        const ElemT *b = ptr_b + n * stride_b;
        for (int k = 0; k < type_length; ++k) {
            val += element_inner_product(a[k], b[k]);
        }
        return val;
    }
};

template<typename ElemT, typename ScalarT>
void array_inner_device(const ElemT *ptr_a, const ElemT *ptr_b, ScalarT *ptr_out, int count, int byte_stride_a,
                        int byte_stride_b, int type_length) {
    assert((byte_stride_a % sizeof(ElemT)) == 0);
    assert((byte_stride_b % sizeof(ElemT)) == 0);
    const int stride_a = byte_stride_a / sizeof(ElemT);
    const int stride_b = byte_stride_b / sizeof(ElemT);

    void *context = cuda_context_get_current();
    TemporaryBuffer &cub_temp = g_temp_buffer_map[context];

    ContextGuard guard(context);
    auto stream = static_cast<hipStream_t>(cuda_stream_get_current());

    cub_inner_product_iterator<ElemT, ScalarT> inner_iterator{ptr_a, ptr_b, stride_a, stride_b, type_length};

    size_t buff_size = 0;
    check_cuda(hipcub::DeviceReduce::Sum(nullptr, buff_size, inner_iterator, ptr_out, count, stream));
    cub_temp.ensure_fits(buff_size);

    check_cuda(hipcub::DeviceReduce::Sum(cub_temp.buffer, buff_size, inner_iterator, ptr_out, count, stream));
}

template<typename T>
void array_inner_device_dispatch(const T *ptr_a, const T *ptr_b, T *ptr_out, int count, int byte_stride_a,
                                 int byte_stride_b, int type_length) {
    using vec2 = wp::vec_t<2, T>;
    using vec3 = wp::vec_t<3, T>;
    using vec4 = wp::vec_t<4, T>;

    // specialized calls for common vector types

    if ((type_length % 4) == 0 && (byte_stride_a % sizeof(vec4)) == 0 && (byte_stride_b % sizeof(vec4)) == 0) {
        return array_inner_device(reinterpret_cast<const vec4 *>(ptr_a), reinterpret_cast<const vec4 *>(ptr_b), ptr_out,
                                  count, byte_stride_a, byte_stride_b, type_length / 4);
    }

    if ((type_length % 3) == 0 && (byte_stride_a % sizeof(vec3)) == 0 && (byte_stride_b % sizeof(vec3)) == 0) {
        return array_inner_device(reinterpret_cast<const vec3 *>(ptr_a), reinterpret_cast<const vec3 *>(ptr_b), ptr_out,
                                  count, byte_stride_a, byte_stride_b, type_length / 3);
    }

    if ((type_length % 2) == 0 && (byte_stride_a % sizeof(vec2)) == 0 && (byte_stride_b % sizeof(vec2)) == 0) {
        return array_inner_device(reinterpret_cast<const vec2 *>(ptr_a), reinterpret_cast<const vec2 *>(ptr_b), ptr_out,
                                  count, byte_stride_a, byte_stride_b, type_length / 2);
    }

    return array_inner_device(ptr_a, ptr_b, ptr_out, count, byte_stride_a, byte_stride_b, type_length);
}

}// anonymous namespace

void array_inner_float_device(uint64_t a, uint64_t b, uint64_t out, int count, int byte_stride_a, int byte_stride_b,
                              int type_len) {
    void *context = cuda_context_get_current();

    const auto *ptr_a = (const float *)(a);
    const auto *ptr_b = (const float *)(b);
    auto *ptr_out = (float *)(out);

    array_inner_device_dispatch(ptr_a, ptr_b, ptr_out, count, byte_stride_a, byte_stride_b, type_len);
}

void array_inner_double_device(uint64_t a, uint64_t b, uint64_t out, int count, int byte_stride_a, int byte_stride_b,
                               int type_len) {
    const auto *ptr_a = (const double *)(a);
    const auto *ptr_b = (const double *)(b);
    auto *ptr_out = (double *)(out);

    array_inner_device_dispatch(ptr_a, ptr_b, ptr_out, count, byte_stride_a, byte_stride_b, type_len);
}

void array_sum_float_device(uint64_t a, uint64_t out, int count, int byte_stride, int type_length) {
    const auto *ptr_a = (const float *)(a);
    auto *ptr_out = (float *)(out);
    array_sum_device_dispatch(ptr_a, ptr_out, count, byte_stride, type_length);
}

void array_sum_double_device(uint64_t a, uint64_t out, int count, int byte_stride, int type_length) {
    const auto *ptr_a = (const double *)(a);
    auto *ptr_out = (double *)(out);
    array_sum_device_dispatch(ptr_a, ptr_out, count, byte_stride, type_length);
}
